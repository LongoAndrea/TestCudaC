#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string.h>

#include <stdio.h>

#include "CSha.h"

#define SIZE_X 17576
#define SIZE_Y 4


using std::string;
using std::endl;
using std::cout;

typedef unsigned char BYTE;

hipError_t memAlloc(BYTE *matrix);

__global__ void getCombinations(BYTE *matrix) 
{
	for (int i = 0; i < SIZE_X; i++)
		for (int j = 0; j < SIZE_Y; j++)
			matrix[i + j * SIZE_X] = 1;
}

int main()
{
	BYTE host_matrix[SIZE_X * SIZE_Y];

	for (int i = 0; i < SIZE_X; i++)
		for (int j = 0; j < SIZE_Y; j++)
			host_matrix[i + j * SIZE_X] = 0;

	memAlloc(host_matrix);

	return 0;
}

hipError_t memAlloc(BYTE *matrix) 
{
	BYTE *dev_matrix;

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_matrix, SIZE_X * SIZE_Y * sizeof(BYTE));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_matrix);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_matrix, matrix, SIZE_X * SIZE_Y * sizeof(BYTE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_matrix);
		return cudaStatus;
	}


	getCombinations <<< 1, SIZE_Y >>> (dev_matrix);
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_matrix);
		return cudaStatus;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		hipFree(dev_matrix);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(matrix, dev_matrix, SIZE_X * SIZE_Y * sizeof(BYTE), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_matrix);
		return cudaStatus;
	}

	hipFree(dev_matrix);
	return cudaStatus;
}